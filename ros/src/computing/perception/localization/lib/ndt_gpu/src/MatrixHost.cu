#include "hip/hip_runtime.h"
#include "ndt_gpu/MatrixHost.h"
#include "ndt_gpu/debug.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

namespace gpu {

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols>::MatrixHost()
{
	is_copied_ = false;
	buffer_ = NULL;
	offset_ = 0;
}

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols>::MatrixHost(int offset, Scalar *buffer) :
Matrix<Scalar, Rows, Cols>(offset, buffer)
{}

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols>::MatrixHost(const MatrixHost<Scalar, Rows, Cols>& other) {
	if (rows_ > 0 && cols_ > 0) {
		offset_ = other.offset_;
		is_copied_ = true;
		rows_ = other.rows_;
		cols_ = other.cols_;
		buffer_ = other.buffer_;
	}
}

template <typename Scalar, int Rows, int Cols>
__global__ void copyMatrixDevToDev(MatrixDevice<Scalar, Rows, Cols> input, MatrixDevice<Scalar, Rows, Cols> output) {
	int row = threadIdx.x;
	int col = threadIdx.y;

	if (row < input.rows() && col < input.cols())
		output(row, col) = input(row, col);
}

template <typename Scalar, int Rows, int Cols>
bool MatrixHost<Scalar, Rows, Cols>::moveToGpu(MatrixDevice<Scalar, Rows, Cols> output) {
	if (rows_ != output.rows() || cols_ != output.cols())
		return false;

	Scalar *tmp;

	checkCudaErrors(hipMalloc(&tmp, sizeof(Scalar) * rows_ * cols_ * offset_));
	checkCudaErrors(hipMemcpy(tmp, buffer_, sizeof(Scalar) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));

	MatrixDevice<Scalar, Rows, Cols> tmp_output(rows_, cols_, offset_, tmp);

	dim3 block_x(rows_, cols_, 1);
	dim3 grid_x(1, 1, 1);

	copyMatrixDevToDev<Scalar, Rows, Cols><<<grid_x, block_x>>>(tmp_output, output);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(tmp));

	return true;
}

template <typename Scalar, int Rows, int Cols>
bool MatrixHost<Scalar, Rows, Cols>::moveToHost(const MatrixDevice<Scalar, Rows, Cols> input) {
	if (rows_ != input.rows() || cols_ != input.cols())
		return false;

	Scalar *tmp;

	checkCudaErrors(hipMalloc(&tmp, sizeof(Scalar) * rows_ * cols_ * offset_));

	MatrixDevice<Scalar, Rows, Cols> tmp_output(rows_, cols_, offset_, tmp);

	dim3 block_x(rows_, cols_, 1);
	dim3 grid_x(1, 1, 1);

	copyMatrixDevToDev<Scalar, Rows, Cols><<<grid_x, block_x>>>(input, tmp_output);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(buffer_, tmp, sizeof(Scalar) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(tmp));

	return true;

}

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols> &MatrixHost<Scalar, Rows, Cols>::operator=(const MatrixHost<Scalar, Rows, Cols> &other)
{
	buffer_ = other.buffer_;
	offset_ = other.offset_;
	is_copied_ = true;
	rows_ = other.rows_;
	cols_ = other.cols_;

	return *this;
}

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols> &MatrixHost<Scalar, Rows, Cols>::operator=(MatrixHost<Scalar, Rows, Cols> &&other)
{
	if (!is_copied_ && buffer_ != NULL) {
		free(buffer_);
		is_copied_ = false;
	}

	offset_ = other.offset_;
	is_copied_ = false;
	buffer_ = other.buffer_;
	rows_ = other.rows_;
	cols_ = other.cols_;

	other.is_copied_ = true;
	other.buffer_ = NULL;

	return *this;
}


template <typename Scalar, int Rows, int Cols>
void MatrixHost<Scalar, Rows, Cols>::debug()
{
	std::cout << *this;
}

template <typename Scalar, int Rows, int Cols>
std::ostream &operator<<(std::ostream &os, const MatrixHost<Scalar, Rows, Cols> &value)
{
	for (int i = 0; i < value.rows(); i++) {
		for (int j = 0; j < value.cols(); j++) {
			os << value.at(i, j) << " ";
		}

		os << std::endl;
	}

	os << std::endl;

	return os;
}

template <typename Scalar, int Rows, int Cols>
MatrixHost<Scalar, Rows, Cols>::~MatrixHost()
{
	if (!is_copied_ && buffer_ != NULL)
		free(buffer_);
}

template class MatrixHost<float>;
template class MatrixHost<double>;

}
